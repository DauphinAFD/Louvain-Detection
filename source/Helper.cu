#include "hip/hip_runtime.h"
#include "../include/HELPER.cuh"

using namespace std;

bool isPrime(int n) {
	for (int i = 2; i < sqrt(n) + 1; i++)
		if (n % i == 0)
			return false;
	return true;
}

int findPrime(int n) {
    n++;
	while(!isPrime(n)) {
		n++;
	} 
	return n;
}

int LargestDegree(host_graph& Hgraph) {
    int maxDeg = 0;

    for (int i = 0; i < Hgraph.V; i++){
        
        maxDeg = std::max(maxDeg, Hgraph.edgesIdx[i+1] - Hgraph.edgesIdx[i]);
        
    }
    
    return maxDeg;
}

host_graph Read(string filename){

    int V, E;
    int v, u, idx = 0;
    string line;
    float weight;
    stringstream ss;
    host_graph Hgraph;
    fstream louvain(filename);
    if(louvain.is_open()){
        
        getline(louvain,line);
        ss.clear();
        ss.str("");
        ss.str(line);
        ss >> V >> E;
        // cout << "Original :" << V << E << endl;
        
        Hgraph.old_V = V;
        Hgraph.V = V;
        Hgraph.E = E;

        size_t Cwghtsize = V*sizeof(float);
        size_t idxsize = (V+1)*sizeof(int);
        size_t graphsize = V*sizeof(int);
        Hgraph.commWeights = (float *)malloc(Cwghtsize);
        Hgraph.oldToNewComm = (int *)malloc(graphsize);
        Hgraph.vtxComm = (int *)malloc(graphsize);
        Hgraph.edgesIdx = (int *)malloc(idxsize);

        vector<vector<pair<int , float>>> neigh(V);
        thrust::fill(thrust::host, Hgraph.commWeights, Hgraph.commWeights + V, 0);
        while(getline(louvain,line)) {
            ss.clear();
            ss.str("");
            ss.str(line);
        
            ss >> v >> u >> weight;
            //cout << v << " " << u << " " << weight << endl;
            Hgraph.commWeights[v] += weight;
            neigh[v].push_back(make_pair(u, weight));
            Hgraph.M += weight;
            if(v != u){
                E++;
                Hgraph.commWeights[u] += weight;
                neigh[u].push_back(make_pair(v, weight));
                Hgraph.M += weight;
            }
        }

        Hgraph.M /= 2.0;
        // cout << Hgraph.M << endl;
        Hgraph.E = E;
        size_t wghtsize = E*sizeof(float);
        size_t edgesize = E*sizeof(int);
        Hgraph.weights = (float *)malloc(wghtsize);
        Hgraph.edges = (int *)malloc(edgesize);
       

        for(int i = 0; i < V; i++){
           
            Hgraph.edgesIdx[i] = idx;
            for(auto x : neigh[i]){
                Hgraph.edges[idx] = x.first;
                Hgraph.weights[idx] = x.second;
                
                idx++;
            }
        }
        Hgraph.edgesIdx[V] = Hgraph.E;

        louvain.close();
        
    }
    
    return Hgraph;


}

void prepareDevice(host_graph &Hgraph, device_graph & Dgraph, new_graph &NewGraph){
    int V = Hgraph.V;
    int E = Hgraph.E;

    //cout << "Entering Function"<<endl;

    size_t Cwghtsize = V*sizeof(float);
    size_t idxsize = (V+1)*sizeof(int);
    size_t wghtsize = E*sizeof(float);
    size_t graphsize = V*sizeof(int);
    size_t edgesize = E*sizeof(int);

    CHECK(hipMalloc((void**)&Dgraph.commWeights, Cwghtsize));
    CHECK(hipMalloc((void**)&Dgraph.newVtxComm, graphsize));
    CHECK(hipMalloc((void**)&Dgraph.vtx_Kvalue, Cwghtsize));
    CHECK(hipMalloc((void**)&Dgraph.partition, graphsize));
    CHECK(hipMalloc((void**)&Dgraph.oldToNewComm, graphsize));
    CHECK(hipMalloc((void**)&Dgraph.weightSum_InComm, graphsize));
    CHECK(hipMalloc((void**)&Dgraph.CommSize, graphsize));
    CHECK(hipMalloc((void**)&Dgraph.vtxComm, graphsize));
    CHECK(hipMalloc((void**)&Dgraph.old_V,sizeof(int)));
    CHECK(hipMalloc((void**)&Dgraph.weights,wghtsize));
    CHECK(hipMalloc((void**)&Dgraph.edgesIdx,idxsize));
    CHECK(hipMalloc((void**)&Dgraph.edges, edgesize));
    CHECK(hipMalloc((void**)&Dgraph.V,sizeof(int)));
    CHECK(hipMalloc((void**)&Dgraph.E,sizeof(int)));

	thrust::sequence(thrust::device, Dgraph.newVtxComm, Dgraph.newVtxComm + V, 0);
	thrust::sequence(thrust::device, Dgraph.oldToNewComm, Dgraph.oldToNewComm + V, 0);
	thrust::sequence(thrust::device, Dgraph.vtxComm, Dgraph.vtxComm + V, 0);
    thrust::fill(thrust::device, Dgraph.CommSize, Dgraph.CommSize + V, 1);

    CHECK(hipMemcpy(Dgraph.commWeights, Hgraph.commWeights, Cwghtsize, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(Dgraph.weights, Hgraph.weights, wghtsize, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(Dgraph.edgesIdx, Hgraph.edgesIdx, idxsize, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(Dgraph.old_V, &Hgraph.old_V, sizeof(int), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(Dgraph.edges, Hgraph.edges, edgesize, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(Dgraph.V, &Hgraph.V, sizeof(int), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(Dgraph.E, &Hgraph.E, sizeof(int), hipMemcpyHostToDevice));

	CHECK(hipMalloc((void**)&NewGraph.edgeTocurPos, edgesize));
	CHECK(hipMalloc((void**)&NewGraph.newWeights, wghtsize));
	CHECK(hipMalloc((void**)&NewGraph.orderCom, graphsize));
	CHECK(hipMalloc((void**)&NewGraph.VtxStart, graphsize));
	CHECK(hipMalloc((void**)&NewGraph.newEdges, edgesize));
    CHECK(hipMalloc((void**)&NewGraph.commDeg, graphsize));
	CHECK(hipMalloc((void**)&NewGraph.edgePos, graphsize));
	CHECK(hipMalloc((void**)&NewGraph.newId, graphsize));

    //cout << "finish Preparing"<<endl;

}

void deleteDeviceVars(host_graph &Hgraph, device_graph & Dgraph, new_graph &NewGraph){

    //cout << "Started deleting" << endl;

    CHECK(hipFree(Dgraph.weightSum_InComm));
	CHECK(hipFree(Dgraph.oldToNewComm));
	CHECK(hipFree(Dgraph.commWeights));
	CHECK(hipFree(Dgraph.vtx_Kvalue));
	CHECK(hipFree(Dgraph.newVtxComm));
	CHECK(hipFree(Dgraph.partition));
	CHECK(hipFree(Dgraph.CommSize));
	CHECK(hipFree(Dgraph.edgesIdx));
    CHECK(hipFree(Dgraph.vtxComm));
	CHECK(hipFree(Dgraph.weights));
	CHECK(hipFree(Dgraph.edges));
	CHECK(hipFree(Dgraph.old_V));
	CHECK(hipFree(Dgraph.E));
	CHECK(hipFree(Dgraph.V));
    
	CHECK(hipFree(NewGraph.edgeTocurPos));
	CHECK(hipFree(NewGraph.newWeights));
	CHECK(hipFree(NewGraph.orderCom));
	CHECK(hipFree(NewGraph.VtxStart));
	CHECK(hipFree(NewGraph.newEdges));
	CHECK(hipFree(NewGraph.edgePos));
	CHECK(hipFree(NewGraph.commDeg));
	CHECK(hipFree(NewGraph.newId));
    
    free(Hgraph.oldToNewComm);
    free(Hgraph.commWeights);
    free(Hgraph.edgesIdx);
    free(Hgraph.vtxComm);
    free(Hgraph.weights);
    free(Hgraph.edges);

    //cout << "finish deleting" << endl;
    
}
